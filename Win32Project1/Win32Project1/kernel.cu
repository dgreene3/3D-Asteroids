#include "hip/hip_runtime.h"
#pragma once
#include "kernel.h"
#include <math.h>
#include <glm\glm.hpp>


#ifndef PI
#define PI 3.14159265358979323846f
#endif


using namespace std;

__global__ void kernel(float3* pos, unsigned int width, unsigned int height, float thetaStep, float phiStep) {
	//NOTE: threadIdx.x is row and blockIdx.x is col

	float x, y, z;
	//get index and current offset into memeory
	int index = width * threadIdx.y + threadIdx.x;

	// bottom row of mesh grid
	if(threadIdx.y == 0) {
		x = 0.0f;
		y = -1.0f;
		z = 0.0f;
	}
	// top row of mesh grid
	else if(threadIdx.y == height-1) {
		x = 0.0f;
		y = 1.0f;
		z = 0.0f;
	}else {
		//used to calculate postion
		float theta, phi;
		theta = (float)(PI - (threadIdx.y * thetaStep));
		phi = (float)(threadIdx.x * phiStep);

		//used to calculate spherical position
		x = 1.0f * sinf(theta) * cosf(phi);
		y = 1.0f * cosf(theta);
		z = 1.0f * sinf(theta) * sinf(phi);
	}
	
	//use index to get correct place in array
	pos[index] = make_float3(x, y, z);
}





__global__ void kernel2(unsigned int* i, unsigned int width, unsigned int height) {
	unsigned int index = width * threadIdx.y + threadIdx.x;

	unsigned int offset = index * 6; // every thread/index will compute 6 vertices/2 triangles

	i[offset] = index;
	i[offset+1] = index+1;
	i[offset+2] = index+width;

	i[offset+3] = index+1;
	i[offset+4] = index+1+width;
	i[offset+5] = index+width;
}



__global__ void kernel3(float2* tex, unsigned int width, unsigned int height, float dx, float dy) {
	int index = width * threadIdx.y + threadIdx.x;

	float u = dx * threadIdx.x; // texture coordinate for X left right
	float v = 1.0f - (dy * threadIdx.y); /* DevIL requires this to be flipped? */
	//float v = dy * threadIdx.y;

	tex[index] = make_float2(u, v);

}


//__global__ void kernel4(float3* pos, unsigned int width, unsigned int height) {
//	float x, y, z;
//	//get index and current offset into memeory
//	int index = width * threadIdx.y + threadIdx.x;
//
//	float3 position = pos[index];
//	glm::vec3 temp = glm::normalize(glm::vec3(pos->x, pos->y, pos->z));
//	temp = glm::normalize(temp);
//	/* Look in monte carlo sim example for using hiprand on GPU */
//
//	//use index to get correct place in array
//	pos[index] = make_float3(x, y, z);
//}



void kernel_Pos(float3* pos, unsigned int width, unsigned int height, float thetaFac, float phiFac) {
	dim3 block, grid; /* block describes threads within a block; grid describes how many blocks overall */

	kernel<<< dim3(1, 1, 1), dim3(width, height, 1)>>>(pos, width, height, thetaFac, phiFac);
}

void kernel_Tex(float2* tex, unsigned int width, unsigned int height, float dx, float dy) {
	dim3 block, grid; /* block describes threads within a block; grid describes how many blocks overall */

	kernel3<<< dim3(1, 1, 1), dim3(width, height, 1)>>>(tex, width, height, dx, dy);
}



void kernel_Index(unsigned int* i, unsigned int width, unsigned int height) {

	kernel2<<<dim3(1, 1, 1), dim3((width-1), (height-1), 1) >>>(i, width, height); /* width-1 */
}


//void kernel_Asteroid(float3* pos, unsigned int width, unsigned int height) {
//	kernel4<<< dim3(1, 1, 1), dim3(width, height, 1)>>>(pos, width, height);
//}